#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "grid.h"

__host__ __device__ bool getAt(Grid *grid, int i, int j)
{
    return grid->getAt(i, j);
}

__host__ __device__ void setAt(Grid *grid, int i, int j, bool value)
{
    grid->setAt(i, j, value);
}

__host__ __device__ int getWidth(Grid *grid)
{
    return grid->getWidth();
}

__host__ __device__ int getHeight(Grid *grid)
{
    return grid->getHeight();
}

// Kernel
__global__ void computeHighLife(Grid *grid, Grid *result)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < getWidth(grid) and j < getHeight(grid) and i >= 0 and j >= 0)
    {
        setAt(result, i, j, !getAt(grid, i, j));
    }
}

// Cuda main
extern "C"
int cuda_main(Grid *grid)
{
    Grid *mygrid = new Grid(grid->getWidth(), grid->getHeight());
    Grid *result = new Grid(grid->getWidth(), grid->getHeight());

    *mygrid = *grid;
    *result = *grid;

    hipMallocManaged(&mygrid, sizeof(mygrid));
    hipMallocManaged(&result, sizeof(result));

    int blocksize = 32;
    dim3 threads(blocksize, blocksize);
    dim3 cudagrid(mygrid->getWidth() / threads.x, mygrid->getHeight() / threads.y);

    std::cout << "CUDA can receive a Grid object" << std::endl;
    std::cout << &mygrid << std::endl;
    std::cout << "mygrid.getAt(0, 0) was = " << mygrid->getAt(0, 0) << std::endl; // SEGFAULT
    std::cout << "result.getAt(0, 0) was = " << result->getAt(0, 0) << std::endl;
    computeHighLife<<< cudagrid, threads >>>(mygrid, result);
    std::cout << "CUDA can send a Grid object" << std::endl;
    std::cout << "mygrid.getAt(0, 0) is = " << std::boolalpha << mygrid->getAt(0, 0) << std::endl;
    std::cout << "result.getAt(0, 0) is = " << std::boolalpha << result->getAt(0, 0) << std::endl;

    *mygrid = *result;
    return 0;
}
