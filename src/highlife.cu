#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "grid.h"
#include "stdio.h"

// Helper 2D -> 1D array
__host__ __device__ int getPos(int i, int j, int n)
{
    return i + n * j;
}

// Kernel
__global__ void computeHighLife(bool **grid, bool *result, int width, int height)
{
//     int i = blockDim.x * blockIdx.x + threadIdx.x;
//     int j = blockDim.y * blockIdx.y + threadIdx.y;

//     if (grid[threadIdx.x % height][threadIdx.x / height] and not (surroundingAliveCells(i, j) == 2 or surroundingAliveCells(i, j) == 3))
//     {
        //!(grid[threadIdx.x][threadIdx.y]);
        result[getPos(threadIdx.x, threadIdx.y, width)] = (threadIdx.y >= threadIdx.x);
//     }

//     if (i < getWidth(grid) and j < getHeight(grid) and i >= 0 and j >= 0)
//     {
//         setAt(result, i, j, !getAt(grid, i, j));
//     }
}

// Cuda main
extern "C"
int cuda_main(Grid *grid)
{
    bool **h_grid = grid->getInnerGrid();
    bool *h_result = (bool *)malloc(grid->getWidth() * grid->getHeight() * sizeof(bool));
    bool **d_grid;
    hipMalloc(&d_grid, grid->getWidth() * grid->getHeight() * sizeof(bool));
    bool *d_result;
    hipMalloc(&d_result, grid->getWidth() * grid->getHeight() * sizeof(bool));

    h_grid = grid->getInnerGrid();

    for (int j = 0; j < grid->getHeight(); j++)
    {
        for (int i = 0; i < grid->getWidth(); i++)
        {
            h_result[getPos(i, j, grid->getWidth())] = 1;
        }
    }

    std::cout << "Host listo" << std::endl;

    // Copy vectors from host memory to device memory
    hipMemcpy(d_grid, h_grid, grid->getWidth() * grid->getHeight() * sizeof(bool), hipMemcpyHostToDevice);

    dim3 gridSize(grid->getWidth(), grid->getHeight());

    std::cout << "CUDA can receive a Grid object?" << std::endl;

    computeHighLife<<< 1, gridSize >>>(d_grid, d_result, grid->getWidth(), grid->getHeight());

    // h_result contains the result in host memory
    hipMemcpy(h_result, d_result, grid->getWidth() * grid->getHeight() * sizeof(bool), hipMemcpyDeviceToHost);

    std::cout << "CUDA can send a Grid object?" << std::endl;

    for (int j = 0; j < grid->getHeight(); j++)
    {
        for (int i = 0; i < grid->getWidth(); i++)
        {
            grid->setAt(i, j, h_result[getPos(i, j, grid->getWidth())]);
        }
    }

    // Final result
    return 0;
}

/*

OLD IDEA

__host__ __device__ bool getAt(Grid *grid, int i, int j)
{
    return grid->getAt(i, j);
}

__host__ __device__ void setAt(Grid *grid, int i, int j, bool value)
{
    grid->setAt(i, j, value);
}

__host__ __device__ int getWidth(Grid *grid)
{
    return grid->getWidth();
}

__host__ __device__ int getHeight(Grid *grid)
{
    return grid->getHeight();
}

// Kernel
__global__ void computeHighLife(Grid *grid, Grid *result)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    setAt(result, 1, 0, false);
//     if (i < getWidth(grid) and j < getHeight(grid) and i >= 0 and j >= 0)
//     {
//         setAt(result, i, j, !getAt(grid, i, j));
//     }
}

// Cuda main
extern "C"
int cuda_main(Grid *grid)
{
    Grid *mygrid;
    Grid *result;

    hipMallocManaged(&mygrid, sizeof(mygrid));
    hipMallocManaged(&result, sizeof(result));

    mygrid = new Grid(grid->getWidth(), grid->getHeight());
    result = new Grid(grid->getWidth(), grid->getHeight());

    *mygrid = *grid;
    *result = *grid;

    int blocksize = 32;
    dim3 threads(blocksize, blocksize);
    dim3 cudagrid(mygrid->getWidth() / threads.x, mygrid->getHeight() / threads.y);

    // FIXME Cuda puede hacer modificaciones (TODO), pero hay que ponerle ojo a los margenes, o hace segfault
    std::cout << "CUDA can receive a Grid object" << std::endl;
    std::cout << &mygrid << std::endl;
    std::cout << "mygrid.getAt(1, 0) was = " << std::boolalpha << mygrid->getAt(1, 0) << std::endl;
    std::cout << "result.getAt(1, 0) was = " << std::boolalpha << result->getAt(1, 0) << std::endl;
    computeHighLife<<< cudagrid, threads >>>(mygrid, result);
    std::cout << "CUDA can send a Grid object" << std::endl;
    std::cout << "mygrid.getAt(1, 0) is = " << std::boolalpha << mygrid->getAt(1, 0) << std::endl;
    std::cout << "result.getAt(1, 0) is = " << std::boolalpha << result->getAt(1, 0) << std::endl;

    // Final result
    *grid = *result;
    return 0;
}
*/
